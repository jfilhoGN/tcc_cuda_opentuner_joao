#include <hip/hip_runtime.h>
#include <stdio.h>
// #include "dimensions.h"

__global__ void checkIndex(int funcId) {
  /*printf("threadIdx:(%d, %d, %d) blockIdx:(%d, %d, %d) blockDim:(%d, %d, %d) "
         "gridDim:(%d, %d, %d) -> id: %d\n",
         threadIdx.x, threadIdx.y, threadIdx.z, blockIdx.x, blockIdx.y,
         blockIdx.z, blockDim.x, blockDim.y, blockDim.z, gridDim.x, gridDim.y,
         gridDim.z, getGlobalIdFunc[funcId]());*/
    printf("threadIdx:(%d, %d, %d) blockIdx:(%d, %d, %d) blockDim:(%d, %d, %d) "
         "gridDim:(%d, %d, %d) -> id: %d\n",
         threadIdx.x, threadIdx.y, threadIdx.z, blockIdx.x, blockIdx.y,
         blockIdx.z, blockDim.x, blockDim.y, blockDim.z, gridDim.x, gridDim.y,
         gridDim.z);
}

int main(int argc, char **argv) {

  if (argc != 9) {
        printf("Uso: %s <g.x> <g.y> <g.z> <b.x> <b.y> <b.z> <funcId> <gpuId>\n", argv[0]);
        printf("     funcId:\n");
        printf("     0: 1D_1D, 1: 1D_2D, 2: 1D_3D\n");
        printf("     3: 2D_1D, 4: 2D_2D, 5: 2D_3D\n");
        printf("     6: 3D_1D, 7: 3D_2D, 8: 3D_3D\n");
        return 0;
    }
  /* Definição do arranjo de threads em blocos do grid. */
  dim3 grid(atoi(argv[1]), atoi(argv[2]), atoi(argv[3]));
  dim3 block(atoi(argv[4]), atoi(argv[5]), atoi(argv[6]));

  int funcId = atoi(argv[7]);
  int gpuId =  atoi(argv[8]);

  /* Define the gpu id to work */
  hipSetDevice(gpuId);

  // check grid and block dimension from host side
  printf("grid.x %d grid.y %d grid.z %d\n", grid.x, grid.y, grid.z);
  printf("block.x %d block.y %d block.z %d\n", block.x, block.y, block.z);
  
  // check grid and block dimension from device side
  checkIndex<<<grid, block>>>(funcId);
  
  // reset device before you leave
  hipDeviceReset();
  return (0);
}
