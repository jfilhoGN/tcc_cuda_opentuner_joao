#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <malloc.h>
 
#ifndef DATA_TYPE
#define DATA_TYPE float
#endif

DATA_TYPE N = 1024;

__device__ int getGlobalIdx_1D_1D() {
    // Operações -> multiply: 1 add: 1 (2 FLOPs).
    // printf("getGlobalIdx_1D_1D.\n");
    return blockIdx.x * blockDim.x + threadIdx.x;
}
__device__ int getGlobalIdx_1D_2D() {
    // Operações -> multiply: 3 add: 2 (5 FLOPs).
    // printf("getGlobalIdx_1D_2D.\n");
    return blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x
            + threadIdx.x;
}
__device__ int getGlobalIdx_1D_3D() {
    // Operações -> multiply: 6 add: 3 (9 FLOPs).
    // printf("getGlobalIdx_1D_3D.\n");
    return blockIdx.x * blockDim.x * blockDim.y * blockDim.z
            + threadIdx.z * blockDim.y * blockDim.x + threadIdx.y * blockDim.x
            + threadIdx.x;
}
__device__ int getGlobalIdx_2D_1D() {
    // Operações -> multiply: 2 add: 2 (4 FLOPs).
    // printf("getGlobalIdx_2D_1D.\n");
    int blockId = blockIdx.y * gridDim.x + blockIdx.x;
    int threadId = blockId * blockDim.x + threadIdx.x;
    return threadId;
}
__device__ int getGlobalIdx_2D_2D() {
    // Operações -> multiply: 4 add: 3 (7 FLOPs).
    // printf("getGlobalIdx_2D_2D.\n");
    int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    int threadId = blockId * (blockDim.x * blockDim.y)
            + (threadIdx.y * blockDim.x) + threadIdx.x;
    return threadId;
}
__device__ int getGlobalIdx_2D_3D() {
    // Operações -> multiply: 7 add: 4 (11 FLOPs).
    // printf("getGlobalIdx_2D_3D.\n");
    int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
    + (threadIdx.z * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x)
            + threadIdx.x;
    return threadId;
}
__device__ int getGlobalIdx_3D_1D() {
    // Operações -> multiply: 4 add: 3 (7 FLOPs).
    // printf("getGlobalIdx_3D_1D.\n");
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
            + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * blockDim.x + threadIdx.x;
    return threadId;
}
__device__ int getGlobalIdx_3D_2D() {
    // Operações -> multiply: 6 add: 4 (10 FLOPs).
    // printf("getGlobalIdx_3D_2D.\n");
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
            + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y)
            + (threadIdx.y * blockDim.x) + threadIdx.x;
    return threadId;
}
__device__ int getGlobalIdx_3D_3D() {
    // Operações -> multiply: 9 add: 5 (14 FLOPs).
    // printf("getGlobalIdx_3D_3D.\n");
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
            + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
            + (threadIdx.z * (blockDim.x * blockDim.y))
            + (threadIdx.y * blockDim.x) + threadIdx.x;
    return threadId;
}

__global__ void vecAdd(DATA_TYPE *a, DATA_TYPE *b, DATA_TYPE *c, int n){
    //Thread ID
    int id = blockIdx.x*blockDim.x+threadIdx.x;
 
    if (id < n)
        c[id] = a[id] + b[id];
}
 
int main(int argc, char **argv){
 	
 	int blockSize, gridSize;
 	//Tamanho do block size será passado pelo opentuner
    // Numero de threads por block
    blockSize = atoi(argv[1]);

    // Host entrada vetor
    int *h_a;
    int *h_b;

    //Host saida vetor
    int *h_c;
 
    // Device entrada vetor
    int *d_a;
    int *d_b;

    //Device saida vetor
    int *d_c;
 
    // Tamanho, em bytes
    size_t bytes = N*sizeof(int);
 
    // Alocando o tamanho do vetor
    h_a = (int*)malloc(bytes);
    h_b = (int*)malloc(bytes);
    h_c = (int*)malloc(bytes);
 
    // Alocando memoria para a GPU
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);
 
    int i;
    // Inicializa vetor e adicionando valores
    for( i = 0; i < N; i++ ) {
        h_a[i] = i;
        h_b[i] = i;
    }
 
    // Copia do vetor Host para o vetor Device
    hipMemcpy( d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy( d_b, h_b, bytes, hipMemcpyHostToDevice);

    // Numero de thread blocks na grid
    gridSize = (int)ceil((float)N/blockSize);
 
    // Executa o kernel
    vecAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c, N);
 
    // Copia array de volta para o host
    hipMemcpy( h_c, d_c, bytes, hipMemcpyDeviceToHost );
 
    for(i=0; i<N; i++){
    	printf("resultado: %d\n", h_c[i]);
    }
    
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
 
    free(h_a);
    free(h_b);
    free(h_c);
 
    return 0;
}
