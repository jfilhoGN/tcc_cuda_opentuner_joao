#include "hip/hip_runtime.h"
/**
 * atax.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <unistd.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "../../dimensions.h"
#include "../common/polybenchUtilFuncts.h"
#include "../common/polybench.c"

#include <hip/hip_runtime.h>

#include "../../../dimensions.h"


//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.5

#define GPU_DEVICE 0

#ifndef M_PI
#define M_PI 3.14159
#endif

/* Can switch DATA_TYPE between float and double */
typedef float DATA_TYPE;

void init_array(DATA_TYPE *x, DATA_TYPE *A, int NX, int NY)
{
	int i, j;

	for (i = 0; i < NX; i++)
	{
		x[i] = i * M_PI;
		for (j = 0; j < NY; j++)
		{
			A[i*NY + j] = ((DATA_TYPE) i*(j)) / NX;
		}
	}
}


void compareResults(DATA_TYPE *z, DATA_TYPE *z_outputFromGpu, int NY)
{
	int i, fail;
	fail = 0;

	for (i=0; i<NY; i++)
	{
		if (percentDiff(z[i], z_outputFromGpu[i]) > PERCENT_DIFF_ERROR_THRESHOLD)
		{
			fail++;
		}		
	}
	
	// print results
	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}


void GPU_argv_init()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
	printf("setting device %d with name %s\n",GPU_DEVICE,deviceProp.name);
	hipSetDevice( GPU_DEVICE );
}


__global__ void atax_kernel1(DATA_TYPE *A, DATA_TYPE *x, DATA_TYPE *tmp, int NX, int NY, funcId)
{
	int i = getGlobalIdFunc[funcId]();

	if (i < NX)
	{
		int j;
		for(j=0; j < NY; j++)
		{
			tmp[i] += A[i * NY + j] * x[j];
		}
	}
}

__global__ void atax_kernel2(DATA_TYPE *A, DATA_TYPE *y, DATA_TYPE *tmp, int NX, int NY, int funcId)
{
	int j = getGlobalIdFunc[funcId]();
	
	if (j < NY)
	{
		int i;
		for(i=0; i < NX; i++)
		{
			y[j] += A[i * NY + j] * tmp[i];
		}
	}
}


void atax_cpu(DATA_TYPE* A, DATA_TYPE* x, DATA_TYPE* y, DATA_TYPE* tmp, int NX, int NY)
{
	int i,j;
	
	for (i= 0; i < NY; i++)
	{
    	y[i] = 0;
	}
  
	for (i = 0; i < NX; i++)
 	{
      	tmp[i] = 0;

      	for (j = 0; j < NY; j++)
		{
			tmp[i] = tmp[i] + A[i*NY + j] * x[j];
		}
		
      	for (j = 0; j < NY; j++)
		{
			y[j] = y[j] + A[i*NY + j] * tmp[i];
		}
    }
}


int main(int argc, char** argv)
{
	double t_start, t_end;

	int NX = 0;
	int NY = 0;
	int kernel = 0;
	int funcId = 0;
	int i = 0;

	if (argc != 11) {
        printf("Uso: %s <kernel> <g.x> <g.y> <g.z> <b.x> <b.y> <b.z> <nx> <ny> \n", argv[0]);
        /*printf("     funcId:\n");
        printf("     0: 1D_1D, 1: 1D_2D, 2: 1D_3D\n");
        printf("     3: 2D_1D, 4: 2D_2D, 5: 2D_3D\n");
        printf("     6: 3D_1D, 7: 3D_2D, 8: 3D_3D\n");*/
        return 0;
    }
    else{
        printf("#argumentos (argc): %d\n", argc);
        for (i = 0; i < argc; ++i) {
           printf(" argv[%d]: %s\n", i, argv[i]);
        }
    
        kernel = atoi(argv[1]);
        NX = atoi(argv[8]);
        NY = atoi(argv[9]);
        //funcId = atoi(argv[10]);
        printf("Executando: %s atax_kernel_%d grid(%d, %d, %d) block(%d, %d, %d) %d\n", argv[0], kernel, atoi(argv[2]), atoi(argv[3]), atoi(argv[4]), atoi(argv[5]), atoi(argv[6]), atoi(argv[7]));
    }
  
    /* Recuperar as informações da GPU. */
    printf("%s Starting...\n", argv[0]);

	DATA_TYPE* A;
	DATA_TYPE* x;
	DATA_TYPE* y;
	DATA_TYPE* y_outputFromGpu;
	DATA_TYPE* tmp;

	A = (DATA_TYPE*)malloc(NX*NY*sizeof(DATA_TYPE));
	x = (DATA_TYPE*)malloc(NY*sizeof(DATA_TYPE));
	y = (DATA_TYPE*)malloc(NY*sizeof(DATA_TYPE));
	y_outputFromGpu = (DATA_TYPE*)malloc(NY*sizeof(DATA_TYPE));
	tmp = (DATA_TYPE*)malloc(NX*sizeof(DATA_TYPE));

	init_array(x, A, NX, NY);

	GPU_argv_init();
	DATA_TYPE *A_gpu;
	DATA_TYPE *x_gpu;
	DATA_TYPE *y_gpu;
	DATA_TYPE *tmp_gpu;

	hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * NX * NY);
	hipMalloc((void **)&x_gpu, sizeof(DATA_TYPE) * NY);
	hipMalloc((void **)&y_gpu, sizeof(DATA_TYPE) * NY);
	hipMalloc((void **)&tmp_gpu, sizeof(DATA_TYPE) * NX);
	
	hipMemcpy(A_gpu, A, sizeof(DATA_TYPE) * NX * NY, hipMemcpyHostToDevice);
	hipMemcpy(x_gpu, x, sizeof(DATA_TYPE) * NY, hipMemcpyHostToDevice);
	hipMemcpy(y_gpu, y, sizeof(DATA_TYPE) * NY, hipMemcpyHostToDevice);
	hipMemcpy(tmp_gpu, tmp, sizeof(DATA_TYPE) * NX, hipMemcpyHostToDevice);
	
	if (kernel == 0){
		dim3 block(atoi(argv[5]), atoi(argv[6]), atoi(argv[7]));
		dim3 grid1(atoi(argv[2]), atoi(argv[3]), atoi(argv[4]));
		funcId = calculateFunctionId(grid, block);
  		printf("funcId: %d\n", funcId);
		t_start = rtclock();
		atax_kernel1<<< grid1, block >>>(A_gpu,x_gpu,tmp_gpu, NX, NY, funcId);
		hipDeviceSynchronize();
		t_end = rtclock();
		fprintf(stdout, "GPU Runtime: %0.6lfs\n", t_end - t_start);
		hipMemcpy(y_outputFromGpu, y_gpu, sizeof(DATA_TYPE) * NX, hipMemcpyDeviceToHost);
		hipFree(A_gpu);
		hipFree(x_gpu);
		hipFree(y_gpu);
		hipFree(tmp_gpu);
	}else{
		dim3 block(atoi(argv[5]), atoi(argv[6]), atoi(argv[7]));
		dim3 grid2(atoi(argv[2]), atoi(argv[3]), atoi(argv[4]));
		funcId = calculateFunctionId(grid, block);
  		printf("funcId: %d\n", funcId);
		t_start = rtclock();
		atax_kernel2<<< grid2, block >>>(A_gpu,y_gpu,tmp_gpu, NX, NY, funcId);
		hipDeviceSynchronize();
		t_end = rtclock();
		fprintf(stdout, "GPU Runtime: %0.6lfs\n", t_end - t_start);
		hipMemcpy(y_outputFromGpu, y_gpu, sizeof(DATA_TYPE) * NX, hipMemcpyDeviceToHost);
		hipFree(A_gpu);
		hipFree(x_gpu);
		hipFree(y_gpu);
		hipFree(tmp_gpu);
	}
	
	t_start = rtclock();
	atax_cpu(A, x, y, tmp, NX, NY);
	t_end = rtclock();
	fprintf(stdout, "CPU Runtime: %0.6lfs\n", t_end - t_start);

	compareResults(y, y_outputFromGpu, NY);

	free(A);
	free(x);
	free(y);
	free(y_outputFromGpu);
	free(tmp);

  	return 0;
}

