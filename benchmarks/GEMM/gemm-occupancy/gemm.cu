#include "hip/hip_runtime.h"
/**
 * gemm.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

 // FALTA SO CHAMAR AS DIMENSOES E ARRUMAR NO KERNEL

#include <unistd.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <stdlib.h>
#include <stdarg.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include "../../../dimensions.h"
#include "../common/polybenchUtilFuncts.h"
#include "../common/polybench.c"


#define GPU_DEVICE 0

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05


/* Declared constant values for ALPHA and BETA (same as values in PolyBench 2.0) */
#define ALPHA 32412.0f
#define BETA 2123.0f

/* Can switch DATA_TYPE between float and double */
typedef float DATA_TYPE;

void gemm(DATA_TYPE *A, DATA_TYPE *B, DATA_TYPE *C, int NI, int NJ, int NK)
{
	int i,j,k;
	
	for (i = 0; i < NI; i++)
	{
    	for (j = 0; j < NJ; j++)
    	{
			C[i*NJ + j] *= BETA;
	
			for (k = 0; k < NK; ++k)
			{
	  			C[i*NJ + j] += ALPHA * A[i*NK + k] * B[k*NJ + j];
			}
      	}
	}
}


void init(DATA_TYPE *A, DATA_TYPE *B, DATA_TYPE *C, int NI, int NJ, int NK)
{
	int i, j;

  	for (i = 0; i < NI; i++)
	{
    	for (j = 0; j < NK; j++)
		{
      		A[i*NK + j] = ((DATA_TYPE) i*j) / NI;
		}
	}

  	for (i = 0; i < NK; i++)
	{
    	for (j = 0; j < NJ; j++)
		{
      		B[i*NJ + j] = ((DATA_TYPE) i*j + 1) / NJ;
		}
	}

  	for (i = 0; i < NI; i++)
	{
    	for (j = 0; j < NJ; j++)
		{
      		C[i*NJ + j] = ((DATA_TYPE) i*j + 2) / NJ;
		}
	}
}


void compareResults(DATA_TYPE* C, DATA_TYPE* C_outputFromGpu, int NI, int NJ)
{
	int i, j, fail;
	fail = 0;
	
	// Compare C1 and C2
	for (i=0; i < NI; i++) 
	{
		for (j=0; j < NJ; j++) 
		{
			if (percentDiff(C[i*NJ + j], C_outputFromGpu[i*NJ + j]) > PERCENT_DIFF_ERROR_THRESHOLD) 
			{
				fail++;
			}
		}
	}
	
	// Print results
	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}


void GPU_argv_init()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
	printf("setting device %d with name %s\n",GPU_DEVICE,deviceProp.name);
	hipSetDevice( GPU_DEVICE );
}


__global__ void gemm_kernel(DATA_TYPE *a, DATA_TYPE *b, DATA_TYPE *c, int NI, int NJ, int NK, int funcId)
{
	int j = getGlobalIdFunc[funcId]();
	int i = getGlobalIdFunc[funcId]();

	if ((i < NI) && (j < NJ))
	{	
		c[i * NJ + j] *= BETA;
		int k;
		for(k=0; k < NK; k++)
		{
			c[i * NJ + j] += ALPHA * a[i * NK + k] * b[k * NJ +j];
		}
	}
}
	

int main(int argc, char *argv[])
{
	double t_start, t_end;

	int NJ = 0;
	int NI = 0;
	int NK = 0;
    int kernel = 0;
    int funcId = 0;
    int i = 0;
    if (argc != 11) {
        printf("Uso: %s <kernel> <g.x> <g.y> <g.z> <b.x> <b.y> <b.z> <ni> <nj> <nk> \n", argv[0]);
        /*printf("     funcId:\n");
        printf("     0: 1D_1D, 1: 1D_2D, 2: 1D_3D\n");
        printf("     3: 2D_1D, 4: 2D_2D, 5: 2D_3D\n");
        printf("     6: 3D_1D, 7: 3D_2D, 8: 3D_3D\n");*/
        return 0;
    }
    else{
        printf("#argumentos (argc): %d\n", argc);
        for (i = 0; i < argc; ++i) {
           printf(" argv[%d]: %s\n", i, argv[i]);
        }
    
        kernel = atoi(argv[1]);
        NI = atoi(argv[8]);
        NJ = atoi(argv[9]);
        NK = atoi(argv[10]);
        //funcId = atoi(argv[11]);
        //printf("Executando: %s gemm_kernel_%d grid(%d, %d, %d) block(%d, %d, %d) %d\n", argv[0], kernel, atoi(argv[2]), atoi(argv[3]), atoi(argv[4]), atoi(argv[5]), atoi(argv[6]), atoi(argv[7]));
    }
  
    /* Recuperar as informações da GPU. */
    printf("%s Starting...\n", argv[0]);

	DATA_TYPE* A;
	DATA_TYPE* B;  
	DATA_TYPE* C;  
	DATA_TYPE* C_outputFromGpu; 

	A = (DATA_TYPE*)malloc(NI*NK*sizeof(DATA_TYPE)); 
	B = (DATA_TYPE*)malloc(NK*NJ*sizeof(DATA_TYPE));   
	C = (DATA_TYPE*)malloc(NI*NJ*sizeof(DATA_TYPE)); 
	C_outputFromGpu = (DATA_TYPE*)malloc(NI*NJ*sizeof(DATA_TYPE)); 


	DATA_TYPE *A_gpu;
	DATA_TYPE *B_gpu;
	DATA_TYPE *C_gpu;

	hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * NI * NK);
	hipMalloc((void **)&B_gpu, sizeof(DATA_TYPE) * NK * NJ);
	hipMalloc((void **)&C_gpu, sizeof(DATA_TYPE) * NI * NJ);

	init(A, B, C, NI, NJ, NK);
	
	GPU_argv_init();
	
	hipMemcpy(A_gpu, A, sizeof(DATA_TYPE) * NI * NK, hipMemcpyHostToDevice);
	hipMemcpy(B_gpu, B, sizeof(DATA_TYPE) * NK * NJ, hipMemcpyHostToDevice);
	hipMemcpy(C_gpu, C, sizeof(DATA_TYPE) * NI * NJ, hipMemcpyHostToDevice);
	
	dim3 block(atoi(argv[5]), atoi(argv[6]), atoi(argv[7]));
	dim3 grid(atoi(argv[2]), atoi(argv[3]), atoi(argv[4]));
	
	if (kernel==0){
		funcId = calculateFunctionId(grid, block);
		t_start = rtclock();
		gemm_kernel<<< grid, block >>>(A_gpu, B_gpu, C_gpu, NI, NJ, NK, funcId);
		hipDeviceSynchronize();

		t_end = rtclock();
		fprintf(stdout, "GPU Runtime: %0.6lfs\n", t_end - t_start);

		hipMemcpy(C_outputFromGpu, C_gpu, sizeof(DATA_TYPE) * NI * NJ, hipMemcpyDeviceToHost);    
	
		hipFree(A_gpu);
		hipFree(B_gpu);
		hipFree(C_gpu);
	}
	

	t_start = rtclock();	
	gemm(A, B, C, NI, NJ, NK);
	t_end = rtclock();
	fprintf(stdout, "CPU Runtime: %0.6lfs\n", t_end - t_start);
	
	compareResults(C, C_outputFromGpu, NI, NJ);

	free(A);
	free(B);  
	free(C);  
	free(C_outputFromGpu); 

    return 0;
}

